
#include <hip/hip_runtime.h>


// This enable the CUDA code to be call from R ( Wrapper function in R creation)

extern "C" void gvectorAdd(double *A, double *B, double *C, int *n);



// This is kernel : executed on the device

__global__ void
vectorAdd(const double *A, const double *B, double *C, int numElements)
{ double A2 = 0  ; double B2 = 0 ; 
  int i = blockDim.x * blockIdx.x + threadIdx.x;
 
  if(i < numElements)
  { 
    A2   = (A[i]*A[i])/numElements ;
    B2   = (B[i]*B[i])/numElements ;

    C[i] = A2 + B2 ;
  }
}




// main code configuration needed to launch the kernel

void gvectorAdd(double *A, double *B, double *C, int *n) 
{
  // Device Memory

  double *d_A, *d_B, *d_C;


  // Define the execution configuration

  double THREADS = 1024;
  
  double n_blocksx = ceil(*n/THREADS); 

  dim3 threadPerBlock(THREADS);
  dim3 numBlocks(n_blocksx);

  // Allocate memory on the device


  hipMalloc((void**)&d_A, *n * sizeof(double));
  hipMalloc((void**)&d_B, *n * sizeof(double));
  hipMalloc((void**)&d_C, *n * sizeof(double));

  // copy data from host to device

  hipMemcpy(d_A, A, *n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, *n * sizeof(double), hipMemcpyHostToDevice);

  // Launching the kernel

  vectorAdd<<<numBlocks,threadPerBlock>>>(d_A, d_B, d_C, *n);
  
  // Copy output from device back to the host
  
  hipMemcpy(C, d_C, *n * sizeof(double), hipMemcpyDeviceToHost);

  // Free device memory 
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

// Compiling the whole using nvcc + creating the shared object 

// nvcc --ptxas-options=-v --compiler-options '-fPIC' -o modvecadd.so --shared modvecadd.cu





